
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#ifndef N
#define N 256
#endif

// Grid boundary conditions
#define RIGHT 1.0
#define LEFT 1.0
#define TOP 1.0
#define BOTTOM 10.0

// precision
#ifdef SINGLE
typedef float REAL;
#define TOLERANCE 0.0001f
 #define POW powf
#define SQRT sqrtf
#else
typedef double REAL;
#define TOLERANCE 0.0001
#define POW pow
#define SQRT sqrt
#endif


// Algorithm settings
#define NPRINT 1000
#define MAX_ITER 200000


__global__
void stencil_sum(REAL*grid, REAL *grid_new, int nx, int ny)
{
  int index=blockIdx.x * blockDim.x +threadIdx.x; // global thread id

  int nrow=index/ny;
  int diff=index-(nrow*ny);
  int k=(nrow+1)*(ny+2)+diff+1;

  if (index<nx*ny) 
      grid_new[k]=REAL(0.25f) * (grid[k-1]+grid[k+1] + grid[k-(ny+2)] + grid[k+(ny+2)]);
}

__global__
void stencil_norm(REAL*grid, REAL*arraynorm, int nx, int ny)
{
  int index=blockIdx.x * blockDim.x +threadIdx.x; // globEl thread id
  
  int nrow=index/ny;
  int diff=index-(nrow*ny);
  int k=(nrow+1)*(ny+2)+diff+1;

  if (index<nx*ny)
     arraynorm[index]=(REAL)POW(grid[k]*REAL(4.0f)-grid[k-1]-grid[k+1] - grid[k-(ny+2)] - grid[k+(ny+2)], 2);

}

//   
//  Taken from CUDA document. Uses  Reduce v4. 
//  Partial sums performed for each block
//  

__global__
void reduce(REAL* g_idata, REAL *g_odata, int nx, int ny) {
extern __shared__ REAL sdata[];

  int tid=threadIdx.x;
  int i=blockIdx.x*(blockDim.x*2) + threadIdx.x;

  if ( (i+blockDim.x) < (nx*ny) ) 
     sdata[tid]=g_idata[i]+g_idata[i+blockDim.x];
  else
     sdata[tid]=0.0;

  __syncthreads();

  for(int s=blockDim.x/2;s>0;s>>=1) {
     if (tid<s) {
        sdata[tid] += sdata[tid+s];
     }
     __syncthreads();
  }
  if (tid ==0) { 
      g_odata[blockIdx.x]=sdata[0];
  }
}


// MAIN LOOP 
int main(int argc, char*argv[]) {

  int k;
  REAL tmpnorm,bnorm,norm;


 // One device
  hipSetDevice(0);

  int nx=N;
  int ny=N;


// GPU threads/block

  int blockSize=256;
  int numBlocks = ((nx*ny)+blockSize-1)/blockSize;

//
// host allocated memory
//

  REAL *grid= (REAL*)malloc(sizeof(REAL)*(nx+2)*(ny+2));
  REAL *grid_new= (REAL*)malloc(sizeof(REAL)*(nx+2)*(ny+2));
  REAL *arraynorm= (REAL*)malloc(sizeof(REAL)*nx*ny);
  REAL*blocknorm=(REAL*)malloc(sizeof(REAL)*numBlocks);

  //
  // Device allocated memory
  //

  REAL *d_grid, *d_grid_new, *d_arraynorm, *d_blocknorm;
  hipMalloc(&d_grid,(nx+2)*(ny+2)*sizeof(REAL));
  hipMalloc(&d_grid_new,(nx+2)*(ny+2)*sizeof(REAL));
  hipMalloc(&d_arraynorm,nx*ny*sizeof(REAL));
  hipMalloc(&d_blocknorm,numBlocks*sizeof(REAL)); 

// shared memory size on GPU 
  int smemsize=blockSize*sizeof(REAL);

  // Initialise Grid boundaries
  int i,j;
  for (i=0;i<ny+2;i++) {
    grid_new[i]=grid[i]=TOP;
    j=(ny+2)*(nx+1)+i;
    grid_new[j]=grid[j]=BOTTOM;
  }
  for (i=1;i<nx+1;i++) {
    j=(ny+2)*i;
    grid_new[j]=grid[j]=LEFT;
    grid_new[j+ny+1]=grid[j+ny+1]=RIGHT;
  }
   
  // Initialise rest of grid
  for (i=1;i<=nx;i++) 
    for (j=1;j<=ny;j++)
      k=(ny+2)*i+j;
      grid_new[k]=grid[k]=0.0;
   
  // initial norm factor
  tmpnorm=0.0;
  for (i=1;i<=nx;i++) {
    for (j=1;j<=ny;j++) {
      k=(ny+2)*i+j;            
      tmpnorm=tmpnorm+(REAL)POW(grid[k]*4.0-grid[k-1]-grid[k+1] - grid[k-(ny+2)] - grid[k+(ny+2)], 2); 
    }
  }
  bnorm=SQRT(tmpnorm);

// copy arrays to device

  hipMemcpy(d_grid,grid,(nx+2)*(ny+2)*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_grid_new,grid_new,(nx+2)*(ny+2)*sizeof(REAL), hipMemcpyHostToDevice);


//    MAIN LOOP 
  int iter;
  for (iter=0; iter<MAX_ITER; iter++) {

    // calculate norm array
    stencil_norm<<<numBlocks,blockSize>>>(d_grid,d_arraynorm,nx,ny); 
    
    // perform reduction
    reduce<<<numBlocks,blockSize,smemsize>>>(d_arraynorm,d_blocknorm,nx,ny);
    hipMemcpy(blocknorm,d_blocknorm,numBlocks*sizeof(REAL),hipMemcpyDeviceToHost);
 
    // sum up temporary block sums
    tmpnorm=0.0;
    for (i=0;i<numBlocks;i++) {
      tmpnorm=tmpnorm+blocknorm[i];
    }
   
    norm=(REAL)SQRT(tmpnorm)/bnorm;

    if (norm < TOLERANCE) break;

    stencil_sum<<<numBlocks,blockSize>>>(d_grid,d_grid_new,nx,ny);

  // Wait for GPU to finish
  hipDeviceSynchronize();

    REAL *temp=d_grid_new;
    d_grid_new=d_grid;
    d_grid=temp;

    //if (iter % NPRINT ==0) printf("Iteration =%d ,Relative norm=%e\n",iter,norm);
  }

  printf("Terminated on %d iterations, Relative Norm=%e \n", iter,norm);
  
//  for (i=0;i<=nx+1;i++) {
//    for (j=0;j<=ny+1;j++){
//     printf("->%lf ",grid[j+i*(ny+2)]);
//    }
//    printf("\n");
//  }


  hipFree(d_grid);
  hipFree(d_grid_new);
  hipFree(d_arraynorm);

  free(grid);
  free(grid_new);
  free(arraynorm);

  return 0;
    

  }
