
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#ifndef LOG2N
#define LOG2N 16
#endif

#define N     (1 << LOG2N)

#define LOG2K  7
#define K     (1<<7)


#ifndef FLOAT
#define FLOAT double
#endif



#define ELEMENTARY_LOG2SIZE 11

__global__ void fwtBatch1Kernel(FLOAT *d_Output, FLOAT *d_Input, int log2N){

    const int    n = 1 << log2N;
    int     stride = n;
    const int base = blockIdx.x << log2N;

    //(2 ** 11) * 4 bytes == 8KB -- maximum s_data[] size for G80
    extern __shared__ FLOAT s_data[];
    FLOAT *d_Src = d_Input  + base;
    FLOAT *d_Dst = d_Output + base;

    for(int pos = threadIdx.x; pos < n; pos += blockDim.x)
        s_data[pos] = d_Src[pos];


    //Do single radix-2 stage if for odd power
    if(log2N & 1){
        __syncthreads();
        stride >>= 1;
        for(int pos = threadIdx.x; pos < n / 2; pos += blockDim.x){
            int lo = pos & (stride - 1);
            int i0 = ((pos - lo) << 1) + lo;
            int i1 = i0 + stride;

            FLOAT t0 = s_data[i0];
            FLOAT t1 = s_data[i1];
            s_data[i0] = t0 + t1;
            s_data[i1] = t0 - t1;
        }
    }

    //Main radix4 stages
    stride >>= 2;
    int pos = threadIdx.x;
    for(; stride >= 1; stride >>= 2){
        __syncthreads();
        int lo = pos & (stride - 1);
        int i0 = ((pos - lo) << 2) + lo;
        int i1 = i0 + stride;
        int i2 = i1 + stride;
        int i3 = i2 + stride;

        FLOAT d0 = s_data[i0];
        FLOAT d1 = s_data[i1];
        FLOAT d2 = s_data[i2];
        FLOAT d3 = s_data[i3];

        FLOAT t;
        t = d0; d0         = d0 + d2; d2         = t - d2;
        t = d1; d1         = d1 + d3; d3         = t - d3;
        t = d0; s_data[i0] = d0 + d1; s_data[i1] = t - d1;
        t = d2; s_data[i2] = d2 + d3; s_data[i3] = t - d3;
    }

    __syncthreads();
    for(int pos = threadIdx.x; pos < n; pos += blockDim.x)
        d_Dst[pos] = s_data[pos];
}

////////////////////////////////////////////////////////////////////////////////
// Single in-global memory radix-4 Fast Walsh Transform pass
// (for strides exceeding elementary vector size)
////////////////////////////////////////////////////////////////////////////////
__global__ void fwtBatch2Kernel(
    FLOAT *d_Output,
    FLOAT *d_Input,
    int stride
){
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int   n = blockDim.x *  gridDim.x * 4;

    FLOAT *d_Src = d_Input  + blockIdx.y * n;
    FLOAT *d_Dst = d_Output + blockIdx.y * n;

    int lo = pos & (stride - 1);
    int i0 = ((pos - lo) << 2) + lo;
    int i1 = i0 + stride;
    int i2 = i1 + stride;
    int i3 = i2 + stride;

    FLOAT d0 = d_Src[i0];
    FLOAT d1 = d_Src[i1];
    FLOAT d2 = d_Src[i2];
    FLOAT d3 = d_Src[i3];

    FLOAT t;
    t = d0; d0        = d0 + d2; d2        = t - d2;
    t = d1; d1        = d1 + d3; d3        = t - d3;
    t = d0; d_Dst[i0] = d0 + d1; d_Dst[i1] = t - d1;
    t = d2; d_Dst[i2] = d2 + d3; d_Dst[i3] = t - d3;
}

////////////////////////////////////////////////////////////////////////////////
// Put everything together: batched Fast Walsh Transform CPU front-end
////////////////////////////////////////////////////////////////////////////////
void fwtBatchGPU(FLOAT *d_Data, int M, int log2N){
    int n = 1 << log2N;
    dim3 grid((1 << log2N) / 1024, M, 1);
    for(; log2N > ELEMENTARY_LOG2SIZE; log2N -= 2, n >>= 2, M <<= 2){
        fwtBatch2Kernel<<<grid, 256>>>(d_Data, d_Data, n / 4);
    }

    fwtBatch1Kernel<<<M, n / 4, n * sizeof(FLOAT)>>>(
        d_Data,
        d_Data,
        log2N
    );
}



////////////////////////////////////////////////////////////////////////////////
// Modulate two arrays
////////////////////////////////////////////////////////////////////////////////
__global__ void modulateKernel(FLOAT *d_A, FLOAT *d_B, int n){
    int        tid = blockIdx.x * blockDim.x + threadIdx.x;
    int numThreads = blockDim.x * gridDim.x;
    FLOAT     rcpN = 1.0f / (FLOAT)n;

    for(int pos = tid; pos < n; pos += numThreads)
        d_A[pos] *= d_B[pos] * rcpN;
}

//Interface to modulateKernel()
void modulateGPU(FLOAT *d_A, FLOAT *d_B, int n){
    modulateKernel<<<128, 256>>>(d_A, d_B, n);
}


int main()
{
	int i;


    FLOAT
        *h_Data,
        *h_Kernel,
        *h_ResultGPU;

    FLOAT
        *d_Data,
        *d_Kernel;

	h_Data      = (FLOAT *)malloc(N*sizeof(FLOAT));
    h_Kernel    = (FLOAT *)malloc(K*sizeof(FLOAT));
	h_ResultGPU = (FLOAT *)malloc(N*sizeof(FLOAT));

    hipMalloc((void **)&d_Kernel, N*sizeof(FLOAT));
    hipMalloc((void **)&d_Data,   N*sizeof(FLOAT)); 

    for (i = 0; i < N; i++)
        h_Data[i] = (FLOAT)rand() / (FLOAT)RAND_MAX;
    for (i = 0; i < K; i++)
        h_Kernel[i] = (FLOAT)rand() / (FLOAT)RAND_MAX;    

    hipMemset(d_Kernel, 0, N*sizeof(FLOAT));
    hipMemcpy(d_Kernel, h_Kernel, K*sizeof(FLOAT), hipMemcpyHostToDevice);
    hipMemcpy(d_Data,   h_Data,   N*sizeof(FLOAT), hipMemcpyHostToDevice);

    fwtBatchGPU(d_Data, 1, LOG2N);
    fwtBatchGPU(d_Kernel, 1, LOG2N);
    modulateGPU(d_Data, d_Kernel, N);
    fwtBatchGPU(d_Data, 1, LOG2N);

    hipMemcpy(h_ResultGPU, d_Data, N*sizeof(FLOAT), hipMemcpyDeviceToHost);

	for(i=0; i<N; i++)
		printf("%.15f,", h_ResultGPU[i]);

	return 0;
}