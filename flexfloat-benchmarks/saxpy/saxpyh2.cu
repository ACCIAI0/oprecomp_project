
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_fp16.h>

#define SIZE 1000000

#define nTPB 256


#define FLOAT half

__global__ void init(int n, half2 *x, half2 *y)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  if (idx < n)
  {
    x[idx] = y[idx] =  __float2half2_rn((float)(idx%15));
  }  
}

__global__ void saxpy(int n, float a, half2 *x, half2 *y)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  if (idx < n)
  {
    half2 a2 = __float2half2_rn(a);
    y[idx] = __hfma2(a2, x[idx], y[idx]);
  }
}

int main(){

  FLOAT *hin, *hout, *din, *dout;
  hin  = (FLOAT *)malloc(SIZE*sizeof(FLOAT));
  hout = (FLOAT *)malloc(SIZE*sizeof(FLOAT));
//  for (int i = 0; i < SIZE; i++) hin[i] = i%15;
//  for (int i = 0; i < SIZE; i++) hout[i] = i%15;
  hipMalloc(&din,  SIZE*sizeof(FLOAT));
  hipMalloc(&dout, SIZE*sizeof(FLOAT));

//  cudaMemcpy(din, hin, SIZE*sizeof(FLOAT), cudaMemcpyHostToDevice);
//  cudaMemcpy(dout, hout, SIZE*sizeof(FLOAT), cudaMemcpyHostToDevice);
  init<<<(SIZE/2+nTPB-1)/nTPB,nTPB>>>(SIZE/2, (half2 *)din, (half2 *)dout);

  int k;
  for(k=0; k<5; ++k)
    saxpy<<<(SIZE/2+nTPB-1)/nTPB,nTPB>>>(SIZE/2, 0.5, (half2 *)din, (half2 *)dout);
  hipMemcpy(hout, dout, SIZE*sizeof(FLOAT), hipMemcpyDeviceToHost);
//  for (int i = 0; i < DSIZE; i++)
  printf("%f ... %f\n", hout[0], hout[SIZE-1]);
  return 0;
}

