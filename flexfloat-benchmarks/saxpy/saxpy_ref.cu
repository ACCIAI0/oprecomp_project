
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_fp16.h>

#define SIZE 1000000

#define nTPB 256

#ifndef FLOAT
#define FLOAT double
#endif

__global__ void saxpy(int n, FLOAT a, FLOAT *x, FLOAT *y)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  if (idx < n)
  {
    y[idx] = a * x[idx] + y[idx];
  }
}

int main()
{

  FLOAT *hin, *hout, *din, *dout;
  hin  = (FLOAT *)malloc(SIZE*sizeof(FLOAT));
  hout = (FLOAT *)malloc(SIZE*sizeof(FLOAT));
  for (int i = 0; i < SIZE; i++) hin[i] = i%15;
  for (int i = 0; i < SIZE; i++) hout[i] = i%15;
  hipMalloc(&din,  SIZE*sizeof(FLOAT));
  hipMalloc(&dout, SIZE*sizeof(FLOAT));
  hipMemcpy(din, hin, SIZE*sizeof(FLOAT), hipMemcpyHostToDevice);
  hipMemcpy(dout, hout, SIZE*sizeof(FLOAT), hipMemcpyHostToDevice);

  int k;
  for(k=0; k<5; ++k)
    saxpy<<<(SIZE+nTPB-1)/nTPB,nTPB>>>(SIZE, 0.5124353, din, dout);
  hipMemcpy(hout, dout, SIZE*sizeof(FLOAT), hipMemcpyDeviceToHost);
  for (int i = 0; i < SIZE; i++)
    printf("%f,", hout[i]);
  return 0;
}

